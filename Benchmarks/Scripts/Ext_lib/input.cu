/*
 * Some basic functions for mtx reading and formating
 * 
 * Author: Petros Anastasiadis(panastas@cslab.ece.ntua.gr) 
 */
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "alloc.h"
#include "input.h"
#include <hip/hip_runtime.h>
#include <cusp/io/matrix_market.h>

int mtx_read1(int ** csrRow, int ** cooCol, double ** cooVal, int * n, int * m, int * n_z, char * name)
{
	cusp::csr_matrix<int, double, cusp::host_memory> matrix;

	// load a matrix stored in MatrixMarket format
	cusp::io::read_matrix_market_file(matrix,name);

	/*save the matrix information*/
	*n = matrix.num_rows;
	*m = matrix.num_cols;
	*n_z = matrix.num_entries;

	hipMallocManaged(csrRow, (*n+1)*sizeof(int));
	hipMallocManaged(cooCol, *n_z*sizeof(int));
	hipMallocManaged(cooVal, *n_z*sizeof(double));
	hipDeviceSynchronize();

	/*copy the elements*/
	//numBytes = (*n + 1) * sizeof(int);
	for (int i=0; i < (*n+1); i++) (*csrRow)[i] = matrix.row_offsets[i] ;
	//hipMemcpy(csrRow, &matrix.row_offsets[0], numBytes,hipMemcpyHostToHost);
	//CudaCheckError();

	//numBytes = *n_z * sizeof(int);
	for (int i=0; i < *n_z; i++) (*cooCol)[i] = matrix.column_indices[i] ;
	//hipMemcpy(cooCol, &matrix.column_indices[0], numBytes,hipMemcpyHostToHost);
	//CudaCheckError();

	//numBytes = *n_z * sizeof(double);
	for (int i=0; i < *n_z; i++) (*cooVal)[i] = matrix.values[i] ;
	//hipMemcpy(cooVal, &matrix.values[0], numBytes,hipMemcpyHostToHost);
	//CudaCheckError();

	return 1;
}

int mtx_read(int ** I, int ** cooCol, double ** cooVal, int * n, int * m, int * n_z, char * name)
{
	
	char c;
	char *type, *format, *var_type, *symmetry, *string=NULL;
	FILE *fp ;
	size_t len=0;
	if ((fp=fopen(name, "r"))==NULL){
		printf("Problem in read pass\n");
		exit(1);
	}
	getline(&string, &len, fp);
	strtok(string," ");
	type = strtok(NULL," ");
	format = strtok(NULL," ");
	var_type = strtok(NULL," ");
	symmetry = strtok(NULL,"\n");
	//printf("type=%s, format=%s, var_type=%s, ", type, format, var_type);
	if (strcmp(type,"matrix")){
		printf("type=%s unsupported...terminating\n\n\n\n\n\n\n\n\n\n\n\n", type);
		exit(1);
	}
	if (strcmp(format,"coordinate") ){
		printf("format=%s unsupported...terminating\n\n\n\n\n\n\n\n\n\n\n\n", format);
		exit(1);
	}
	if (strcmp(var_type,"integer") && strcmp(var_type,"real") && strcmp(var_type,"pattern")){
		printf("Var_type=%s unsupported...terminating\n\n\n\n\n\n\n\n\n\n\n\n", var_type);
		exit(1);
	}
	while((c=getc(fp))=='%') while( (c=getc(fp))!='\n') ; 
	ungetc(c, fp);
	int k, lines = 0, sym_k=0;
	fscanf(fp,"%d %d %d", n, m, &lines);
	//printf("n=%d, m=%d, lines=%d, ", *n, *m, lines);
	
	*n_z = 0;
	if (!strcmp(symmetry,"symmetric")){
		get_nz_symmetric(n_z, name);
		//printf("symmetry=symmetric\n");
	}
	else if (!strcmp(symmetry,"general")) {
		*n_z=lines;
		//printf("symmetry=general\n");
	}
	else {
		printf("Invalid symmetry value:%s\n", symmetry); 
		return 0; 
	}
	//printf("n_z=%d\n", *n_z);
	hipMallocManaged(I, *n_z*sizeof(int));
	hipMallocManaged(cooCol, *n_z*sizeof(int));
	hipMallocManaged(cooVal, *n_z*sizeof(double));
	double dum;
	if ( !*I || !*cooCol || !*cooVal ) return 0;
	
	if (!strcmp(symmetry,"symmetric")){
		for (k = 0; k < lines; k++) {
			if (!strcmp(var_type,"pattern")) {
				fscanf(fp,"%d %d", &((*I)[sym_k]), &((*cooCol)[sym_k]));
				(*cooVal)[sym_k]= 1.0;	
			}
			else {	
				fscanf(fp,"%d %d %lf", &((*I)[sym_k]), &((*cooCol)[sym_k]), &dum);
			 	(*cooVal)[sym_k]=(double) dum;
			}
			(*I)[sym_k]--;
			(*cooCol)[sym_k]--;
			sym_k++;
			if ((*I)[sym_k-1] != (*cooCol)[sym_k-1]) {
				(*I)[sym_k] = (*cooCol)[sym_k-1];
				(*cooCol)[sym_k] = (*I)[sym_k-1];
				(*cooVal)[sym_k] = (*cooVal)[sym_k-1];
				sym_k++;
			}
		}
		if (sym_k!=*n_z){
			printf("Error in symmetric read: sym_k=%d n_z=%d\n", sym_k, *n_z);
			return 0;
		}
	}
	else if (!strcmp(symmetry,"general")) 
	{
		for (k = 0; k < lines; k++){
			if (!strcmp(var_type,"pattern")) {
				fscanf(fp,"%d %d", &((*I)[sym_k]), &((*cooCol)[sym_k]));
				(*cooVal)[sym_k]= 1.0;	
			}
			else {	
				fscanf(fp,"%d %d %lf", &((*I)[sym_k]), &((*cooCol)[sym_k]), &dum);
			 	(*cooVal)[sym_k]=(double) dum;
			}
			(*I)[k]--;
			(*cooCol)[k]--;
		}
	}
	quickSort( *I, *cooCol, *cooVal, 0, *n_z-1);
	fclose(fp);
	return 1;
}


void get_nz_symmetric( int * n_z, char* name)
{
	char c;
	FILE *fp ;
	if ((fp=fopen(name, "r"))==NULL){
		printf("Problem in symmetric read pass\n");
		exit(1);
	}

	while((c=getc(fp))=='%') while( (c=getc(fp))!='\n') ; 
	ungetc(c, fp);
	int k, i, j, n, m, lines;
	double x;
	fscanf(fp,"%d %d %d", &n, &m, &lines);
	for (k = 0; k < lines; k++){
		fscanf(fp,"%d %d %lf", &i, &j, &x);
		(*n_z)++;
		if(i!=j) (*n_z)++;
	}
}


	
void csr_transform(float ** A, int n, int m, int n_z, float  *csrValA, int *csrRowPtrA, int *csrColIndA)
{
	int i,j,k=0;
	for (i = 0; i < n; i++){
		csrRowPtrA[i]=k;
		for (j = 0; j < m; j++){
			if (A[i][j]!=0.0){
				csrValA[k]=A[i][j];
				csrColIndA[k]= j;
				k++;
			}
		}
	}
	csrRowPtrA[i]=k;
	if (k!=n_z) printf("Error at non zeroes: %d\n", k-n_z);
	return;
}

void quickSort( int *a, int * b, double * c, int l, int r)
{
	int j;
	if( l < r ) 
	{	// divide and conquer		
		j = partition( a, b, c, l, r);
		quickSort( a, b, c, l, j-1);
		quickSort( a, b, c, j+1, r);
	}
}



int partition( int *a, int * b, double * c, int l, int r) 
{
	int pivot, i, j, t;
	double t1;
	pivot = a[l];
	i = l; j = r+1;
		
	while(1)
	{
		do ++i; while( a[i] <= pivot && i <= r );
   		do --j; while( a[j] > pivot );
   		if( i >= j ) break;
   		t = a[i]; a[i] = a[j]; a[j] = t;
		t = b[i]; b[i] = b[j]; b[j] = t;
		t1 = c[i]; c[i] = c[j]; c[j] = t1;
   	}
   	t = a[l]; a[l] = a[j]; a[j] = t;
	t = b[l]; b[l] = b[j]; b[j] = t;
	t1 = c[l]; c[l] = c[j]; c[j] = t1;
   	return j;
}









