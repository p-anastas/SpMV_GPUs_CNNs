/*
 *  Some GPU utility functions for SpMV multiplication
 *  Author: Petros Anastasiadis(panastas@cslab.ece.ntua.gr) 
 */ 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gpu_util.h"

const char *gpu_get_errmsg(hipError_t err)
{
    return hipGetErrorString(err);
}

const char *gpu_get_last_errmsg()
{
    return gpu_get_errmsg(hipGetLastError());
}

void cudaCheckErrors(const char * msg)
{
        hipError_t __err = hipGetLastError();
        if (__err != hipSuccess) { 
            printf("\nFatal error: %s (%s)\n", msg, hipGetErrorString(__err));
            exit(1); 
        }
}

void *gpu_alloc(size_t count)
{
	void *ret;
	if (hipMalloc(&ret, count) != hipSuccess) {
		printf("Gpu alloc failed: %s\n", gpu_get_last_errmsg());
		exit(1);
	}
	return ret;
}

void gpu_free(void *gpuptr)
{
    hipFree(gpuptr);
}

int copy_to_gpu(const void *host, void *gpu, size_t count)
{
	if (hipMemcpy(gpu, host, count, hipMemcpyHostToDevice) != hipSuccess){
		printf("Copy to GPU failed: %s\n", gpu_get_last_errmsg());
		exit(1);
	}   
	return 1;
}

int copy_from_gpu(void *host, const void *gpu, size_t count)
{
	if (hipMemcpy(host, gpu, count, hipMemcpyDeviceToHost) != hipSuccess){
		printf("Copy to Host failed: %s\n", gpu_get_last_errmsg());
		exit(1);
	}   
	return 1;
}
double gpu_memory_start_count()
{
	size_t free_byte ;
        size_t total_byte ;

        hipMemGetInfo( &free_byte, &total_byte ) ;
	double free_db = (double)free_byte ;
	double total_db = (double)total_byte ;
	double used_db = total_db - free_db ;
	return used_db/1024.0/1024.0;
}


double gpu_memory_stop_count(double used)
{
	size_t free_byte ;
        size_t total_byte ;

	hipMemGetInfo( &free_byte, &total_byte ) ;
	double free_db = (double)free_byte ;
	double total_db = (double)total_byte ;
	double used_db = total_db - free_db ;
	return used_db/1024.0/1024.0 - used;
}


void gpu_memory_print()
{
	size_t free_byte ;
        size_t total_byte ;
	hipError_t cuda_status;

        cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ){
		printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
		exit(1);
	}
	double free_db = (double)free_byte ;
	double total_db = (double)total_byte ;
	double used_db = total_db - free_db ;

        printf("GPU memory usage: used = %lf, free = %lf MB, total = %lf MB\n", used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}




