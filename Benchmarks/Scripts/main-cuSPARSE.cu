#include "hip/hip_runtime.h"
/*
 * A front-end SparseMatrix-Vector(SMV) multiplication implementation
 * 
 * Author: Petros Anastasiadis(panastas@cslab.ece.ntua.gr) 
 */
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include "Ext_lib/alloc.h"
#include "Ext_lib/dmv.h"
#include "Ext_lib/gpu_util.h"
#include "Ext_lib/timer.h"
#include "Ext_lib/input.h"
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <stdint.h>
#include <inttypes.h>

#define NR_ITER 100



double csecond(void) {

    struct timespec tms;

    if (clock_gettime(CLOCK_REALTIME,&tms)) {
        return (0.0);
    }
    /* seconds, multiplied with 1 million */
    int64_t micros = tms.tv_sec * 1000000;
    /* Add full microseconds */
    micros += tms.tv_nsec/1000;
    /* round up if necessary */
    if (tms.tv_nsec % 1000 >= 500) {
        ++micros;
    }
    return( (double) micros /1000000.0) ;
}

static void check_result(double *test, double *orig, size_t n)
{
	size_t  i_fail = vec_equals(test, orig, n, 0.0001);
	if (!i_fail) printf("Checked, ");
	else printf("FAILED %ld times", i_fail );
}

static void report_results(double timer, int flops, int bytes)
{
	double time = timer/NR_ITER;
	double Gflops = flops/(time*1.e9);
	double Gbytes = bytes/(time*1.e9);
	printf("%lf ms ( %lf Gflops/s %lf Gbytes/s)\n",1000.0*time, Gflops, Gbytes);
}

static void error(const char * msg)
{
	printf("Error: %s\n", msg);
	exit(1);
}

int main(int argc, char **argv)
{
	/* Initializations */
	double alf=1.0;
	double beta=0;
	int n,m, n_z, /* *csrRowPtrA, */ *cooCol, *csrRowPtrA;
	double *cooVal, *x, *y;
	hipsparseHandle_t handle1;
    hipsparseCreate(&handle1);
	hipsparseMatDescr_t descA;
	hipsparseCreateMatDescr(&descA);
	hipsparseSetMatType(descA,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descA,HIPSPARSE_INDEX_BASE_ZERO); 
	double timer;

	/* File Input to COO */
	if (argc < 2) error("Too few Arguments");
	char * name = argv[1];
	FILE *fp;
	if ((fp = fopen(name, "r"))==NULL || (strstr(name, "mtx"))==NULL) error("Invalid File");
	fclose(fp);
	//printf("Serial-CSR version:File=%s, ", name);
	//if(!mtx_read(&csrRowPtrA, &cooCol, &cooVal, &n, &m, &n_z, name)) error("input and/or COO convertion failed");
	if(!mtx_read1(&csrRowPtrA, &cooCol, &cooVal, &n, &m, &n_z, name)) error("input and/or COO convertion failed");		
	
	/* Allocate unified space */	
	//hipMallocManaged(&csrRowPtrA, (n+1)*sizeof(*csrRowPtrA));
	hipMallocManaged(&x, m*sizeof(*x));
	double * y_serial = (double*) calloc(n, sizeof(*y_serial));
	hipMallocManaged(&y, n*sizeof(*y));
	if (!csrRowPtrA || !x || !y || !y_serial) error("Vector Alloc failed");
	hipDeviceSynchronize();
	/* Initialize vectors */
	vec_init_rand(x, m, 1.0);
	vec_init(y_serial, n, 0.0);
	vec_init(y, n, 0.0);

	// Bytes per spmv 
	size_t bytes = 0;
    bytes += 2*sizeof(int) * n;     // row pointer
    bytes += 1*sizeof(int) * n_z;  // column index
    bytes += 2*sizeof(double) * n_z;  // A[i,j] and x[j]
    bytes += 2*sizeof(double) * n;     // y[i] = y[i] + ...

	//FLOPS
	int flops = 2 * n_z ;
	
	/*
	printf("csrRowPtrA=\n");
	for (int i = 0; i < n_z; i++) printf("%d ", csrRowPtrA[i]);
	printf("\ncooCol=\n");
	for (int i = 0; i < n_z; i++) printf("%d ", cooCol[i]);
	printf("\ncooVal=\n");
	for (int i = 0; i < n_z; i++) printf("%lf ", cooVal[i]);
	printf("\nx=\n");
	for(int j = 0 ; j < m ; j++) printf("%lf ",x[j]);
	printf("\n");
	*/
	
	/*
	// Warmup!!! 
	hipsparseXcoo2csr(handle1, csrRowPtrA, n_z, n, csrRowPtrA, HIPSPARSE_INDEX_BASE_ZERO);
	hipDeviceSynchronize();	
	cudaCheckErrors("hipsparseXcoo2csr warmup fail");	

	printf("n=%d, m=%d, n_z=%d, ", n, m, n_z);
	
	// Transform to CSR 
	timer = csecond();
	hipsparseXcoo2csr(handle1, csrRowPtrA, n_z, n, csrRowPtrA, HIPSPARSE_INDEX_BASE_ZERO);
	hipDeviceSynchronize();	
	timer = csecond() - timer;
	cudaCheckErrors("hipsparseXcoo2csr fail");
	printf("transform time= %lf ms, ", 1000.0*timer);
	*/

	/*
	// Execute serial CSR 
	timer = csecond();
	for (size_t i = 0; i < NR_ITER; ++i) dmv_csr(csrRowPtrA, cooCol, cooVal, x, y_serial, n);
	timer = csecond() - timer;
	report_results(timer);
	*/

	
	
	
	printf("File=%s, n=%d, m=%d, n_z=%d\nCuSPARCE-CSR : ",name, n, m, n_z);
		
	// Warmup!!! 
	hipsparseDcsrmv(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, m, n_z, &alf, descA, cooVal, csrRowPtrA, cooCol, x, &beta, y);
	hipDeviceSynchronize();	
	cudaCheckErrors("hipsparseDcsrmv warmup fail");	

	// Kernel launch NR_ITER times 
	timer = csecond();
	//hipProfilerStart();
    for (size_t i = 0; i < NR_ITER; ++i) 
		hipsparseDcsrmv(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, m, n_z, &alf, descA, cooVal, csrRowPtrA, cooCol, x, &beta, y);	
	hipDeviceSynchronize();	
    
	//hipProfilerStop();
   	timer = csecond() - timer;
	cudaCheckErrors("hipsparseDcsrmv fail");	

	report_results(timer,flops,bytes);
	
	
	// Hybrid SpMv 
	
	hipsparseHybMat_t hybA;
	timer = csecond();
	hipsparseCreateHybMat(&hybA);
	// cuSPARSE create Hyb descriptor for A 
	hipsparseDcsr2hyb(handle1, n, m, descA, cooVal, csrRowPtrA, cooCol, hybA, 0, HIPSPARSE_HYB_PARTITION_AUTO);
	timer = csecond() - timer;
	cudaCheckErrors("hipsparseDcsr2hyb fail");	
	printf("CuSPARCE-CSR to HYB : %lf ms\n", 1000.0*timer);
	printf("CuSPARCE-Hybrid : ");
	// Warmup!!! 
	hipsparseDhybmv(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alf, descA, hybA, x, &beta, y);
	hipDeviceSynchronize();	
	cudaCheckErrors("hipsparseDhybmv warmup fail");


	// Kernel launch NR_ITER times 
	timer = csecond();
	//hipProfilerStart();
    for (size_t i = 0; i < NR_ITER; ++i) 
		hipsparseDhybmv(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alf, descA, hybA, x, &beta, y);
	hipDeviceSynchronize();	
    
	//hipProfilerStop();
   	timer = csecond() - timer;
	cudaCheckErrors("cusparseDhybrmv fail");	

	report_results(timer,flops,bytes);
	

	/* Sort cooCol */ /*
	size_t pBufferSizeInBytes = 0;
	hipsparseXcsrsort_bufferSizeExt(handle1, n, m, n_z, gpu_csrRowPtrA, gpu_cooCol, &pBufferSizeInBytes); 
	void *pBuffer = (void *) gpu_alloc(sizeof(char)* pBufferSizeInBytes); 
	int *P		  = (int *)  gpu_alloc(sizeof(int)*n_z); 
	hipsparseCreateIdentityPermutation(handle1, n_z, P);
	hipDeviceSynchronize();
	hipsparseXcsrsort(handle1, n, m, n_z, descA, gpu_csrRowPtrA, gpu_cooCol, P, pBuffer); 
	hipDeviceSynchronize();
	double *gpu_cooVal_S = (double *) gpu_alloc(n_z*sizeof(*gpu_cooVal_S));
	hipsparseDgthr(handle1, n_z, gpu_cooVal, gpu_cooVal_S, P, HIPSPARSE_INDEX_BASE_ZERO);
	hipDeviceSynchronize();
	cudaCheckErrors("csr sort fail");
	gpu_free(pBuffer);
	gpu_free(P);

	
	copy_from_gpu(cooCol, gpu_cooCol, n_z*sizeof(*cooCol));
	copy_from_gpu(cooVal, gpu_cooVal_S, n_z*sizeof(*cooVal));
	printf("cooCol=\n");
	for (int i = 0; i < n_z; i++) printf("%d ", cooCol[i]);
	printf("\ncooVal(S)=\n");
	for (int i = 0; i < n_z; i++) printf("%lf ", cooVal[i]);
	printf("\n");
	

	printf("CuSPARCE-CSR-Sorted version:File=%s, n=%d, m=%d, n_z=%d, ",name, n, m, n_z);

	// Warmpup 
	hipsparseDcsrmv(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, m, n_z, &alf, descA, gpu_cooVal_S, gpu_csrRowPtrA, gpu_cooCol, gpu_x, &beta, gpu_y);	
	hipDeviceSynchronize();
	cudaCheckErrors("hipsparseDcsrmv sorted fail");		
	
	// Kernel launch NR_ITER times 
	timer = csecond();
    for (size_t i = 0; i < NR_ITER; ++i) {
		hipsparseDcsrmv(handle1, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, m, n_z, &alf, descA, gpu_cooVal_S, gpu_csrRowPtrA, gpu_cooCol, gpu_x, &beta, gpu_y);	
		hipDeviceSynchronize();	
    }
   	timer = csecond() - timer;
	cudaCheckErrors("hipsparseDcsrmv sorted fail");	

    // Copy result back to host 
	copy_from_gpu(y, gpu_y, n*sizeof(*y));
	check_result(y, y_serial, n);
	report_results(timer);
	
	
/*
	
	// BSR Initializations
	int nnzb = 0, mb, nb, blockdim;
	hipsparseMatDescr_t descB;
	hipsparseCreateMatDescr(&descB);
	hipsparseSetMatType(descB,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descB,HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW; 

	// BSR blockdim 
	for ( blockdim=3; blockdim < 5; blockdim++ ) {
	nb = (n + blockdim-1)/blockdim; 
	mb = (m + blockdim-1)/blockdim; 
	int *gpu_bsrRowPtr  = (int *) 	 gpu_alloc((nb+1)*sizeof(*gpu_bsrRowPtr));
		
	

	timer = csecond();
	// Get nnzb 
	hipsparseXcsr2bsrNnz(handle1, dir, n, m, descA, csrRowPtrA, cooCol, blockdim, descB, gpu_bsrRowPtr, &nnzb); 
	hipDeviceSynchronize();	
	int 	*gpu_bsrCol = (int *) 	 gpu_alloc(nnzb*sizeof(*gpu_bsrCol));
	double  *gpu_bsrVal = (double *) gpu_alloc((blockdim*blockdim)*nnzb*sizeof(*gpu_bsrVal));

	hipsparseDcsr2bsr(handle1, dir, n, m, descA, cooVal, csrRowPtrA, cooCol, blockdim, descB, gpu_bsrVal, gpu_bsrRowPtr, gpu_bsrCol); 
	hipDeviceSynchronize();
	timer = csecond() - timer;
	cudaCheckErrors("hipsparseXcsr2bsrNnz/hipsparseDcsr2bsr fail");
	
	printf("CuSPARCE-CSR to BSR (BlockDim= %d ): %lf ms\n",blockdim, 1000.0*timer);

	// Final allocations/copies 
	double *xp, *yp;
	hipMallocManaged(&xp, mb * blockdim * sizeof(*xp));
	hipMallocManaged(&yp, nb * blockdim * sizeof(*yp));
	if (!xp || !yp ) error("Unified Alloc failed for bsr");

	vec_init_rand_p(xp, m, mb*blockdim, 1.0);

	printf("CuSPARCE-BSR version(dir=row, BlockDim= %d ): ", blockdim);

	// Warmup 
	hipsparseDbsrmv(handle1, dir, HIPSPARSE_OPERATION_NON_TRANSPOSE, nb, mb, nnzb, &alf, descB, gpu_bsrVal, gpu_bsrRowPtr, gpu_bsrCol, blockdim, xp, &beta, yp);
	hipDeviceSynchronize();
	cudaCheckErrors("hipsparseDbsrmv fail");
	
	// Kernel launch NR_ITER times 
	timer = csecond();
	for (size_t i = 0; i < NR_ITER; ++i) 
		hipsparseDbsrmv(handle1, dir, HIPSPARSE_OPERATION_NON_TRANSPOSE, nb, mb, nnzb, &alf, descB, gpu_bsrVal, gpu_bsrRowPtr, gpu_bsrCol, blockdim, xp, &beta, yp);
	hipDeviceSynchronize();
	timer = csecond() - timer;
	cudaCheckErrors("hipsparseDbsrmv fail");
	report_results(timer,flops,bytes);
	
	gpu_free(gpu_bsrRowPtr);
	gpu_free(gpu_bsrCol);
	gpu_free(gpu_bsrVal);
	gpu_free(yp);
	gpu_free(xp);	

	}
	
	*/

	/* Free resources on unified memory */
	gpu_free(x);
	gpu_free(y);
	gpu_free(y_serial);
	//gpu_free(csrRowPtrA);
	gpu_free(cooCol);
	gpu_free(cooVal);
	gpu_free(csrRowPtrA);

    return 0;
}


